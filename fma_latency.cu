// FMA Latency Measurement Kernels
//
// This file contains various CUDA kernels to measure the latency of
// fused multiply-add (FMA) operations under different execution patterns:
// - Basic latency measurement
// - Interleaved execution (ILP)
// - Non-interleaved execution (sequential chains)

#include <hip/hip_runtime.h>
#include <iostream>

using data_type = float;

// Inline assembly macro to read GPU cycle counter
#define clock_cycle() \
    ({ \
        unsigned long long ret; \
        asm volatile("mov.u64 %0, %%clock64;" : "=l"(ret)); \
        ret; \
    })

////////////////////////////////////////////////////////////////////////////////
// Basic FMA Latency

__global__ void
fma_latency(data_type *n, unsigned long long *d_start, unsigned long long *d_end, int *num_fmas) {
    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();
    data_type x = *n;
    data_type y0 = 0;
    data_type y1 = 0;
    data_type y2 = 0;
    data_type y3 = 0;
    data_type y4 = 0;
    data_type y5 = 0;
    data_type y6 = 0;
    data_type y7 = 0;
    data_type y8 = 0;
    // Memory fence to ensure that the reads are done.
    __threadfence();
    start_time = clock_cycle();

    y0 += x * 4; // 1
    y1 += y0 * 4; // 2
    y2 += y1 * 4; // 3
    y3 += y2 * 4; // 4
    y4 += y3 * 4; // 5
    y5 += y4 * 4; // 6
    y6 += y5 * 4; // 7
    y7 += y6 * 4; // 8
    y8 += y7 * 4; // 9
    x += y8 * 4; // 10

    end_time = clock_cycle();

    *n = x;
    *d_start = start_time;
    *d_end =  end_time;
    *num_fmas = 10;
}

////////////////////////////////////////////////////////////////////////////////
// FMA Latency + Instruction Level Parallelism (Interleaved)

__global__ void fma_latency_interleaved(
        data_type *n,
        unsigned long long *d_start,
        unsigned long long *d_end,
        int *num_fmas) {
    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();

    data_type x = *n;
    data_type y = *n;

    data_type tx0 = 0;
    data_type tx1 = 0;
    data_type tx2 = 0;
    data_type tx3 = 0;
    data_type ty0 = 0;
    data_type ty1 = 0;
    data_type ty2 = 0;
    data_type ty3 = 0;

    // Memory fence to ensure that the reads are done.
    __threadfence();

    start_time = clock_cycle();

    tx0 += x * 4; // 1
    ty0 += y * 4; // 2
    tx1 += tx0 * 4; // 3
    ty1 += ty0 * 4; // 4
    tx2 += tx1 * 4; // 5
    ty2 += ty1 * 4; // 6
    tx3 += tx2 * 4; // 7
    ty3 += ty2 * 4; // 8
    x += tx3 * 4; // 9
    y += ty3 * 4; // 10

    end_time = clock_cycle();

    *n = x + y;
    *d_start = start_time;
    *d_end = end_time;
    *num_fmas = 10;
}

////////////////////////////////////////////////////////////////////////////////
// FMA Latency + Sequential Execution (No Interleaving)

__global__ void fma_latency_no_interleave(
    data_type *n,
    unsigned long long *d_start,
    unsigned long long *d_end,
    int *num_fmas) {

    unsigned long long start_time = clock_cycle();
    unsigned long long end_time = clock_cycle();

    data_type x = *n;
    data_type y = *n;
    // Memory fence to ensure that the reads are done.
    __threadfence();

    start_time = clock_cycle();

    /// <--- /your code here --->

    end_time = clock_cycle();

    *n = x + y;
    *d_start = start_time;
    *d_end = end_time;
    *num_fmas = 1;
}

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

// CUDA error checking macro
#define CUDA_CHECK(x) \
    do { \
        hipError_t err = (x); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error " << static_cast<int>(err) << " (" \
                      << hipGetErrorString(err) << ") at " << __FILE__ << ":" \
                      << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

// Macro to run kernel and print timing results.
#define run_kernel_and_print(kernel, d_n, d_start, d_end, num_fmas) \
    do { \
        unsigned long long h_time_start = 0ull, h_time_end = 0ull; \
        data_type result = 0.0f; \
        int h_num_fmas = 0; \
\
        kernel<<<1, 1>>>(d_n, d_start, d_end, num_fmas); \
        CUDA_CHECK(hipDeviceSynchronize()); \
        CUDA_CHECK(hipMemcpy( \
            &h_time_start, \
            d_start, \
            sizeof(unsigned long long), \
            hipMemcpyDeviceToHost)); \
        CUDA_CHECK(hipMemcpy( \
            &h_time_end, \
            d_end, \
            sizeof(unsigned long long), \
            hipMemcpyDeviceToHost)); \
        CUDA_CHECK(hipMemcpy(&result, d_n, sizeof(data_type), hipMemcpyDeviceToHost)); \
        CUDA_CHECK(hipMemcpy(&h_num_fmas, num_fmas, sizeof(int), hipMemcpyDeviceToHost)); \
\
        float duration = static_cast<float>(h_time_end - h_time_start); \
        duration /= h_num_fmas; \
        std::cout << "Latency of " << #kernel \
                  << " code snippet = " << duration << " cycles" \
                  << " (" << h_num_fmas << " FMAs)" \
                  << std::endl; \
    } while (0)

int main() {
    data_type *d_n = nullptr;
    unsigned long long *d_start = nullptr;
    unsigned long long *d_end = nullptr;
    int *num_fmas = nullptr;

    data_type host_val = 4.0f;
    unsigned long long host_init_time = 0ull;
    int host_num_fmas = 0;

    CUDA_CHECK(hipMalloc(&d_n, sizeof(data_type)));
    CUDA_CHECK(hipMalloc(&d_start, sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_end, sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&num_fmas, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_n, &host_val, sizeof(data_type), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        d_start,
        &host_init_time,
        sizeof(unsigned long long),
        hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        d_end,
        &host_init_time,
        sizeof(unsigned long long),
        hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(num_fmas, &host_num_fmas, sizeof(int), hipMemcpyHostToDevice));

    run_kernel_and_print(fma_latency, d_n, d_start, d_end, num_fmas);
    run_kernel_and_print(fma_latency_interleaved, d_n, d_start, d_end, num_fmas);
    run_kernel_and_print(fma_latency_no_interleave, d_n, d_start, d_end, num_fmas);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_n));
    CUDA_CHECK(hipFree(d_start));
    CUDA_CHECK(hipFree(d_end));
    CUDA_CHECK(hipFree(num_fmas));

    return 0;
}